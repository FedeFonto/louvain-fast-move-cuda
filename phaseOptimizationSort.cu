#include "hip/hip_runtime.h"
#ifndef OPTIMIZATION_SORT_CU     
#define OPTIMIZATION_SORT_CU

#include "phaseOptimization.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__
static void select_self_node(
	unsigned int n_nodes,
	unsigned int* source,
	unsigned int* dest,
	unsigned int* community,
	float* value,
	float* self
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_nodes) {
		int node = source[id];
		if (community[node] == dest[id]) {
			self[node] = value[id];
		}
	}
};


void OptimizationPhase::optimize_fast() {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
	hipEvent_t  round_start, copy, transform, reduce_transform;
	hipEventCreate(&round_start);
	hipEventCreate(&copy);
	hipEventCreate(&transform);
	hipEventCreate(&reduce_transform);
	float milliseconds = 0;
	unsigned analyzed = 0;
	float copy_sum = 0;
	float transform_sum = 0;
	float reduce_transform_sum = 0;
	float total = 0;
#endif

	int limit_round;
	int round = 0;

	while (round < community.graph.edge_destination.size()) {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
		hipEventRecord(round_start);
#endif
		key_node_source.resize(STEP_ROUND);
		key_community_dest.resize(STEP_ROUND);
		values_weight.resize(STEP_ROUND);

		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin()));
		auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin(), values_weight.begin()));

		int n_edge_in_buckets;

		auto p = thrust::copy_if(
			thrust::make_transform_iterator(community.start + round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_transform_iterator(community.start + limit_round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_zip_iterator(thrust::make_tuple(community.graph.edge_source.begin() + round, community.graph.edge_destination.begin() + round)),
			selected_edge,
			TestTupleValue(thrust::raw_pointer_cast(neighboorhood_change.data()))
		);

		n_edge_in_buckets = p - selected_edge;

		key_node_source.resize(n_edge_in_buckets);
		key_community_dest.resize(n_edge_in_buckets);
		values_weight.resize(n_edge_in_buckets);

		if (n_edge_in_buckets == 0) {
#if PRINT_DEBUG_LOG
			printf("No elements!\n");
#endif 
			return;
		}


#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		hipEventElapsedTime(&milliseconds, round_start, copy);

		analyzed += n_edge_in_buckets;
		copy_sum += milliseconds;
#endif

		auto self_community = thrust::device_vector<float>(community.graph.n_nodes, 0);

		thrust::transform(
			selected_edge,
			selected_edge + n_edge_in_buckets,
			values_weight.begin(),
			DeltaModularitySort(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, copy, transform);
		transform_sum += milliseconds;
#endif

		/*for (int i = 0; i < n_reduced_edges; i++) {
			std::cout << reduced_key_source[i] << " " << community.communities[reduced_key_source[i]] << " " << reduced_key_dest[i] << " " << reduced_value[i] << std::endl;
		}*/

		auto community_value_pair_input = thrust::make_zip_iterator(thrust::make_tuple(key_community_dest.begin(), values_weight.begin()));
		auto community_value_pair_output = thrust::make_zip_iterator(thrust::make_tuple(final_community.begin() + nodes_considered, final_value.begin() + nodes_considered));

		auto ne = thrust::reduce_by_key(
			key_node_source.begin(),
			key_node_source.begin() + n_edge_in_buckets,
			community_value_pair_input,
			final_node.begin() + nodes_considered,
			community_value_pair_output,
			thrust::equal_to<int>(),
			GetMaxValue()
		);

		nodes_considered = ne.first - final_node.begin();
		round = limit_round;
#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
		hipEventRecord(reduce_transform);
		hipEventSynchronize(reduce_transform);
		hipEventElapsedTime(&milliseconds, transform, reduce_transform);
		reduce_transform_sum += milliseconds;
		hipEventElapsedTime(&milliseconds, round_start, reduce_transform);
		total += milliseconds;
#endif
	}

#if  PRINT_PERFORMANCE_LOG && INCLUDE_FAST
	std::cout << community.graph.edge_destination.size() << ",";
	std::cout << analyzed << ",";
	std::cout << (float)(analyzed) / community.graph.edge_destination.size() << ",";
	std::cout << copy_sum << ",";
	std::cout << transform_sum << ",";
	std::cout << reduce_transform_sum << ",";
	std::cout << total << std::endl;
#endif
}


void OptimizationPhase::optimize_sort() {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
	hipEvent_t  round_start, copy, sort, reduce_sort, self_c, transform, reduce_transform;
	hipEventCreate(&round_start);
	hipEventCreate(&copy);
	hipEventCreate(&sort);
	hipEventCreate(&reduce_sort);
	hipEventCreate(&self_c);
	hipEventCreate(&transform);
	hipEventCreate(&reduce_transform);
	float milliseconds = 0;
	unsigned analyzed = 0;
	float copy_sum = 0;
	float sort_sum = 0;
	float reduce_sort_sum = 0;
	float self_c_sum = 0;
	float transform_sum = 0;
	float reduce_transform_sum = 0;
	float total = 0;
#endif

	int limit_round;
	int round = 0;


	while (round < community.graph.edge_destination.size()) {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(round_start);
#endif
		key_node_source.resize(STEP_ROUND);
		key_community_dest.resize(STEP_ROUND);
		values_weight.resize(STEP_ROUND);

		reduced_key_source.resize(STEP_ROUND);
		reduced_key_dest.resize(STEP_ROUND);
		reduced_value.resize(STEP_ROUND);

		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin()));
		auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin(), values_weight.begin()));

		int n_edge_in_buckets;

		auto p = thrust::copy_if(
			thrust::make_transform_iterator(community.start + round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_transform_iterator(community.start + limit_round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_zip_iterator(thrust::make_tuple(community.graph.edge_source.begin() + round, community.graph.edge_destination.begin() + round)),
			selected_edge,
			TestTupleValue(thrust::raw_pointer_cast(neighboorhood_change.data()))
		);

		n_edge_in_buckets = p - selected_edge;

		key_node_source.resize(n_edge_in_buckets);
		key_community_dest.resize(n_edge_in_buckets);
		values_weight.resize(n_edge_in_buckets);


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		hipEventElapsedTime(&milliseconds, round_start, copy);
		
		analyzed += n_edge_in_buckets;
		copy_sum += milliseconds;
#endif

		if (n_edge_in_buckets == 0) {
#if PRINT_DEBUG_LOG
			printf("No elements!\n");
#endif 
			return;
		}


		int step = 5000000;
		int i = 0;
		for (int off = 0; off < n_edge_in_buckets; off += step) {
			int limit = off + step;
			if (limit >= n_edge_in_buckets) {
				limit = n_edge_in_buckets;
			}
			else {
				limit += community.graph.n_of_neighboor[key_node_source[limit]];
				if (limit >= n_edge_in_buckets) {
					limit = n_edge_in_buckets;
				}
			}
			thrust::sort_by_key(
				key_community + off,
				key_community + limit,
				values_weight.begin() + off
			);
			i++;
		}



#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(sort);
		hipEventSynchronize(sort);
		hipEventElapsedTime(&milliseconds, copy, sort);
		sort_sum += milliseconds;
#endif

		auto reduced_key = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin()));
		auto reduced_list = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin(), reduced_value.begin()));

		auto new_end = thrust::reduce_by_key(
			key_community,
			key_community + n_edge_in_buckets,
			values_weight.begin(),
			reduced_key,
			reduced_value.begin()
		);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_sort);
		hipEventSynchronize(reduce_sort);
		hipEventElapsedTime(&milliseconds, sort, reduce_sort);
		reduce_sort_sum += milliseconds;
#endif

		auto n_reduced_edges = new_end.first - reduced_key;
		reduced_key_source.resize(n_reduced_edges);
		reduced_key_dest.resize(n_reduced_edges);
		reduced_value.resize(n_reduced_edges);

		auto self_community = thrust::device_vector<float>(community.graph.n_nodes);

		int n_blocks = (n_reduced_edges + BLOCK_SIZE - 1) / BLOCK_SIZE;


		select_self_node << <n_blocks, BLOCK_SIZE >> > (
			n_reduced_edges,
			thrust::raw_pointer_cast(reduced_key_source.data()),
			thrust::raw_pointer_cast(reduced_key_dest.data()),
			thrust::raw_pointer_cast(community.communities.data()),
			thrust::raw_pointer_cast(reduced_value.data()),
			thrust::raw_pointer_cast(self_community.data())
			);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(self_c);
		hipEventSynchronize(self_c);
		hipEventElapsedTime(&milliseconds, reduce_sort, self_c);
		self_c_sum += milliseconds;
#endif

		thrust::transform(
			reduced_list,
			reduced_list + n_reduced_edges,
			reduced_value.begin(),
			DeltaModularitySort(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);

		/*for (int i = 0; i < n_reduced_edges; i++) {
			std::cout << reduced_key_source[i] << " " << community.communities[reduced_key_source[i]] << " " << reduced_key_dest[i] << " " << reduced_value[i] << std::endl;
		}*/

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, self_c, transform);
		transform_sum += milliseconds;
#endif

		auto community_value_pair_input = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_dest.begin(), reduced_value.begin()));
		auto community_value_pair_output = thrust::make_zip_iterator(thrust::make_tuple(final_community.begin() + nodes_considered, final_value.begin() + nodes_considered));

		auto ne = thrust::reduce_by_key(
			reduced_key_source.begin(),
			reduced_key_source.begin() + n_reduced_edges,
			community_value_pair_input,
			final_node.begin() + nodes_considered,
			community_value_pair_output,
			thrust::equal_to<int>(),
			GetMaxValue()
		);

		nodes_considered = ne.first - final_node.begin();
		round = limit_round;

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_transform);
		hipEventSynchronize(reduce_transform);
		hipEventElapsedTime(&milliseconds, transform, reduce_transform);
		reduce_transform_sum += milliseconds;
		hipEventElapsedTime(&milliseconds, round_start, reduce_transform);
		total += milliseconds;
#endif
	}

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
	std::cout << community.graph.edge_destination.size() << ",";
	std::cout << analyzed << ",";
	std::cout << (float)(analyzed) / community.graph.edge_destination.size() << ",";
	std::cout << copy_sum << ",";
	std::cout << sort_sum << ",";
	std::cout << reduce_sort_sum << ",";
	std::cout << self_c_sum << ",";
	std::cout << transform_sum << ",";
	std::cout << reduce_transform_sum << ",";
	std::cout << total << std::endl;
#endif
}



#endif