#include "hip/hip_runtime.h"
#ifndef OPTIMIZATION_SORT_CU     
#define OPTIMIZATION_SORT_CU

#include "phaseOptimization.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__
static void select_self_node(
	unsigned int n_nodes,
	unsigned int* source,
	unsigned int* dest,
	unsigned int* community,
	float* value,
	float* self
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_nodes) {
		int node = source[id];
		if (community[node] == dest[id]) {
			self[node] = value[id];
		}
	}
};


void OptimizationPhase::optimize_fast() {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
	hipEvent_t  round_start, copy, sort, reduce_sort, self_c, transform, reduce_transform;
	hipEventCreate(&round_start);
	hipEventCreate(&copy);
	hipEventCreate(&sort);
	hipEventCreate(&reduce_sort);
	hipEventCreate(&self_c);
	hipEventCreate(&transform);
	hipEventCreate(&reduce_transform);
	float milliseconds = 0;
#endif

	int limit_round;
	int round = 0;


	while (round < community.graph.edge_destination.size()) {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(round_start);
#endif
		key_node_source.resize(STEP_ROUND);
		key_community_dest.resize(STEP_ROUND);
		values_weight.resize(STEP_ROUND);

		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin()));
		auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin(), values_weight.begin()));

		int n_edge_in_buckets;

		auto p = thrust::copy_if(
			thrust::make_transform_iterator(community.start + round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_transform_iterator(community.start + limit_round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_zip_iterator(thrust::make_tuple(community.graph.edge_source.begin() + round, community.graph.edge_destination.begin() + round)),
			selected_edge,
			TestTupleValue(thrust::raw_pointer_cast(neighboorhood_change.data()))
		);

		n_edge_in_buckets = p - selected_edge;

		key_node_source.resize(n_edge_in_buckets);
		key_community_dest.resize(n_edge_in_buckets);
		values_weight.resize(n_edge_in_buckets);


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		hipEventElapsedTime(&milliseconds, round_start, copy);
#if CSV_FORM
		std::cout << n_edge_in_buckets << "," << community.graph.edge_source.size() << "," << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << ",";
		std::cout << milliseconds << ",";
#else
		std::cout << "\nNumber of Edges selected: " << n_edge_in_buckets << " / " << community.graph.edge_source.size() << " (" << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << " %)" << std::endl;
		std::cout << " - Copy Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		if (n_edge_in_buckets == 0) {
#if PRINT_DEBUG_LOG
			printf("No elements!\n");
#endif 
			return;
		}


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(sort);
		hipEventSynchronize(sort);
		hipEventElapsedTime(&milliseconds, copy, sort);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Sort Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_sort);
		hipEventSynchronize(reduce_sort);
		hipEventElapsedTime(&milliseconds, sort, reduce_sort);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Reduce after sort time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		auto self_community = thrust::device_vector<float>(community.graph.n_nodes, 0);


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(self_c);
		hipEventSynchronize(self_c);
		hipEventElapsedTime(&milliseconds, reduce_sort, self_c);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Obtain self communities : " << milliseconds << "ms" << std::endl;
#endif
#endif

		thrust::transform(
			selected_edge,
			selected_edge + n_edge_in_buckets,
			values_weight.begin(),
			DeltaModularitySort(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);

		/*for (int i = 0; i < n_reduced_edges; i++) {
			std::cout << reduced_key_source[i] << " " << community.communities[reduced_key_source[i]] << " " << reduced_key_dest[i] << " " << reduced_value[i] << std::endl;
		}*/

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, self_c, transform);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Transform Time : " << milliseconds << "ms" << std::endl;
#endif
#endif



		auto community_value_pair_input = thrust::make_zip_iterator(thrust::make_tuple(key_community_dest.begin(), values_weight.begin()));
		auto community_value_pair_output = thrust::make_zip_iterator(thrust::make_tuple(final_community.begin() + nodes_considered, final_value.begin() + nodes_considered));

		auto ne = thrust::reduce_by_key(
			key_node_source.begin(),
			key_node_source.begin() + n_edge_in_buckets,
			community_value_pair_input,
			final_node.begin() + nodes_considered,
			community_value_pair_output,
			thrust::equal_to<int>(),
			GetMaxValue()
		);

		nodes_considered = ne.first - final_node.begin();
		round = limit_round;

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_transform);
		hipEventSynchronize(reduce_transform);
		hipEventElapsedTime(&milliseconds, transform, reduce_transform);
#if CSV_FORM
		std::cout << milliseconds << "," << std::endl;
#else
		std::cout << " - Reduce after transform Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
	}
}


void OptimizationPhase::optimize_sort() {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
	hipEvent_t  round_start, copy, sort, reduce_sort, self_c, transform, reduce_transform;
	hipEventCreate(&round_start);
	hipEventCreate(&copy);
	hipEventCreate(&sort);
	hipEventCreate(&reduce_sort);
	hipEventCreate(&self_c);
	hipEventCreate(&transform);
	hipEventCreate(&reduce_transform);
	float milliseconds = 0;
#endif

	int limit_round;
	int round = 0;


	while (round < community.graph.edge_destination.size()) {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(round_start);
#endif
		key_node_source.resize(STEP_ROUND);
		key_community_dest.resize(STEP_ROUND);
		values_weight.resize(STEP_ROUND);

		reduced_key_source.resize(STEP_ROUND);
		reduced_key_dest.resize(STEP_ROUND);
		reduced_value.resize(STEP_ROUND);

		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin()));
		auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin(), values_weight.begin()));

		int n_edge_in_buckets;

		auto p = thrust::copy_if(
			thrust::make_transform_iterator(community.start + round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_transform_iterator(community.start + limit_round, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
			thrust::make_zip_iterator(thrust::make_tuple(community.graph.edge_source.begin() + round, community.graph.edge_destination.begin() + round)),
			selected_edge,
			TestTupleValue(thrust::raw_pointer_cast(neighboorhood_change.data()))
		);

		n_edge_in_buckets = p - selected_edge;

		key_node_source.resize(n_edge_in_buckets);
		key_community_dest.resize(n_edge_in_buckets);
		values_weight.resize(n_edge_in_buckets);


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		hipEventElapsedTime(&milliseconds, round_start, copy);
#if CSV_FORM
		std::cout << n_edge_in_buckets << "," << community.graph.edge_source.size() << "," << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << ",";
		std::cout << milliseconds << ",";
#else
		std::cout << "\nNumber of Edges selected: " << n_edge_in_buckets << " / " << community.graph.edge_source.size() << " (" << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << " %)" << std::endl;
		std::cout << " - Copy Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		if (n_edge_in_buckets == 0) {
#if PRINT_DEBUG_LOG
			printf("No elements!\n");
#endif 
			return;
		}


		int step = 5000000;
		int i = 0;
		for (int off = 0; off < n_edge_in_buckets; off += step) {
			int limit = off + step;
			if (limit >= n_edge_in_buckets) {
				limit = n_edge_in_buckets;
			}
			else {
				limit += community.graph.n_of_neighboor[key_node_source[limit]];
				if (limit >= n_edge_in_buckets) {
					limit = n_edge_in_buckets;
				}
			}
			thrust::sort_by_key(
				key_community + off,
				key_community + limit,
				values_weight.begin() + off
			);
			i++;
		}



#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(sort);
		hipEventSynchronize(sort);
		hipEventElapsedTime(&milliseconds, copy, sort);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Sort Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		auto reduced_key = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin()));
		auto reduced_list = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin(), reduced_value.begin()));

		auto new_end = thrust::reduce_by_key(
			key_community,
			key_community + n_edge_in_buckets,
			values_weight.begin(),
			reduced_key,
			reduced_value.begin()
		);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_sort);
		hipEventSynchronize(reduce_sort);
		hipEventElapsedTime(&milliseconds, sort, reduce_sort);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Reduce after sort time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		auto n_reduced_edges = new_end.first - reduced_key;
		reduced_key_source.resize(n_reduced_edges);
		reduced_key_dest.resize(n_reduced_edges);
		reduced_value.resize(n_reduced_edges);

		auto self_community = thrust::device_vector<float>(community.graph.n_nodes);

		int n_blocks = (n_reduced_edges + BLOCK_SIZE - 1) / BLOCK_SIZE;


		select_self_node << <n_blocks, BLOCK_SIZE >> > (
			n_reduced_edges,
			thrust::raw_pointer_cast(reduced_key_source.data()),
			thrust::raw_pointer_cast(reduced_key_dest.data()),
			thrust::raw_pointer_cast(community.communities.data()),
			thrust::raw_pointer_cast(reduced_value.data()),
			thrust::raw_pointer_cast(self_community.data())
			);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(self_c);
		hipEventSynchronize(self_c);
		hipEventElapsedTime(&milliseconds, reduce_sort, self_c);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Obtain self communities : " << milliseconds << "ms" << std::endl;
#endif
#endif

		thrust::transform(
			reduced_list,
			reduced_list + n_reduced_edges,
			reduced_value.begin(),
			DeltaModularitySort(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);

		/*for (int i = 0; i < n_reduced_edges; i++) {
			std::cout << reduced_key_source[i] << " " << community.communities[reduced_key_source[i]] << " " << reduced_key_dest[i] << " " << reduced_value[i] << std::endl;
		}*/

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, self_c, transform);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else
		std::cout << " - Transform Time : " << milliseconds << "ms" << std::endl;
#endif
#endif



		auto community_value_pair_input = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_dest.begin(), reduced_value.begin()));
		auto community_value_pair_output = thrust::make_zip_iterator(thrust::make_tuple(final_community.begin() + nodes_considered, final_value.begin() + nodes_considered));

		auto ne = thrust::reduce_by_key(
			reduced_key_source.begin(),
			reduced_key_source.begin() + n_reduced_edges,
			community_value_pair_input,
			final_node.begin() + nodes_considered,
			community_value_pair_output,
			thrust::equal_to<int>(),
			GetMaxValue()
		);

		nodes_considered = ne.first - final_node.begin();
		round = limit_round;

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(reduce_transform);
		hipEventSynchronize(reduce_transform);
		hipEventElapsedTime(&milliseconds, transform, reduce_transform);
#if CSV_FORM
		std::cout << milliseconds << "," << std::endl;
#else
		std::cout << " - Reduce after transform Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
	}
}



#endif