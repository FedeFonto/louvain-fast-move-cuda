#include "modularityAlgorithms.h"
#include "hip/hip_runtime_api.h"

int main()
{
    std::vector<std::string> dataset = { "soc-pokec-relationships.txt", "soc-LiveJournal.txt", "hollywood-2009.mtx","out.wikipedia_link_it" ,"out.wikipedia_link_fr","com-orkut.ungraph.txt", "out.dbpedia-link" };
    std::vector<int> skip = {0, 0, 49, 1, 1, 0, 2};
   
    //GraphHost g = GraphHost::GraphHost("graph-power-law-huge-2.edge", false, 0);

    for(int i = 0; i< dataset.size(); i++)
    {
        std::cout << std::endl << "****************************** NEW GRAPH ******************************" << std::endl;
        try {
            GraphHost g = GraphHost::GraphHost("dataset/" + dataset[i], false, skip[i]);
            std::cout << std::endl;

            try {
                for (int i = 0; i < 10; i++) {
                    std::cout << std::endl << "################### HASH " << i << " ##################" << std::endl;
                    auto C = ModularityAlgorithms::Louvain(g, HASH);
                    std::cout << "N of community found:" << C.n_of_best_communities << std::endl;
                }
            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl;
            }
            try {
                for (int i = 0; i < 10; i++) {
                    std::cout << std::endl << "################### SORT "<< i << " ##################" << std::endl;
                    auto C = ModularityAlgorithms::Louvain(g, SORT);
                    std::cout << "N of community found:" << C.n_of_best_communities << std::endl;
                }

            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl;
            }

        }
        catch (std::bad_alloc e) {
            std::cout << "Bad Alloc Graph" << std::endl;
            continue;
        }
      
    }


  
    return 0;
}
