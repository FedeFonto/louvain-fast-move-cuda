#include "modularityAlgorithms.h"
#include "hip/hip_runtime_api.h"

int main()
{
    std::vector<std::string> dataset = { "out.patentcite",  "soc-pokec-relationships.txt", "coPapersDBLP.mtx", "packing-500x100x100-b050.mtx","soc-LiveJournal.txt", "ljournal-2008.mtx","out.wikipedia_link_ja",  "hollywood-2009.mtx","out.wikipedia_link_it" , "out.wikipedia_link_fr","com-orkut.ungraph.txt", "out.dbpedia-link", "indochina-2004.mtx" };
    std::vector<int> skip = {2, 0, 129, 141, 0, 56, 1, 49,  1, 1, 0, 2, 48 };

    for(int i = 0; i< dataset.size(); i++)
    {
        std::cout << std::endl << "****************************** NEW GRAPH ******************************" << std::endl;
        try {
            GraphHost g = GraphHost::GraphHost("dataset/" + dataset[i], false, skip[i]);
            std::cout << std::endl;
            Community C = Community(g);

            auto histo_1 = thrust::device_vector <unsigned int>(C.graph.n_of_neighboor.size(), 0);
            auto histo_2 = thrust::device_vector  <int>(C.graph.n_of_neighboor.size(), 0);

            thrust::sort(C.graph.n_of_neighboor.begin(), C.graph.n_of_neighboor.end());
            thrust::reduce_by_key(C.graph.n_of_neighboor.begin(), C.graph.n_of_neighboor.end(), thrust::constant_iterator< int>(1), histo_1.begin(), histo_2.begin());
            auto zip = thrust::make_zip_iterator(thrust::make_tuple(histo_1.begin(), histo_2.begin()));
            auto iii = thrust::make_tuple((unsigned int)0, (int)0);
            auto k = thrust::reduce(zip, zip + C.graph.n_of_neighboor.size(), iii, GetMaxValueint());
            std::cout << thrust::get<0>(k) << " " << thrust::get<1>(k) << std::endl;

        }
        catch (std::bad_alloc e) {
            std::cout << "Bad Alloc Graph" << std::endl;
            continue;
        }
      
    }


  
    return 0;
}
