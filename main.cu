#include "modularityAlgorithms.h"
#include "hip/hip_runtime_api.h"

int main()
{

    GraphHost g = GraphHost::GraphHost("soc-LiveJournal.txt", false);

    hipProfilerStart();
    auto C = ModularityAlgorithms::Laiden(g); 
    hipProfilerStop();
    std::cout<< "N of community found:" << C.n_of_best_communities << std::endl;
    return 0;
}
