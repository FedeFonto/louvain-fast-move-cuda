#include "modularityAlgorithms.h"
#include "hip/hip_runtime_api.h"

int main()
{

    GraphHost g = GraphHost::GraphHost("soc-LiveJournal.txt", false);

    hipProfilerStart();
    auto C = ModularityAlgorithms::Laiden(g); 
    hipProfilerStop();
    return 0;
}
