#include "modularityAlgorithms.h"
#include "hip/hip_runtime_api.h"

int main()
{
    std::vector<std::string> dataset = { "com-orkut.ungraph.txt", "out.trackers", "out.wikipedia_link_fr",  "soc-LiveJournal.txt","hollywood-2009.mtx","out.wikipedia_link_it" ,"soc-pokec-relationships.txt", "out.dbpedia-link" };  
    std::vector<int> skip = { 0, 1, 1, 0, 49,1,0,2};
   
    //GraphHost g = GraphHost::GraphHost("graph-power-law-huge-2.edge", false, 0);

    for(int i = 0; i< dataset.size(); i++)
    {
        std::cout << std::endl << "****************************** NEW GRAPH ******************************" << std::endl;
        try {
            GraphHost g = GraphHost::GraphHost("dataset/" + dataset[i], false, skip[i]);
            std::cout << std::endl;

            try {
                std::cout << std::endl << "################### HASH ##################" << std::endl;
                auto C = ModularityAlgorithms::Louvain(g, HASH);
                std::cout << "N of community found:" << C.n_of_best_communities << std::endl;
            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl;
            }
            try {
                std::cout << std::endl << "################### SORT ##################" << std::endl;
                auto C = ModularityAlgorithms::Louvain(g, SORT);
                std::cout << "N of community found:" << C.n_of_best_communities << std::endl;

            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl;
            }
            try {
                std::cout << std::endl << "################### ADAPTIVE SPEED ##################" << std::endl;
                auto C = ModularityAlgorithms::Louvain(g, ADAPTIVE_SPEED);
                std::cout << "N of community found:" << C.n_of_best_communities << std::endl;
            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl <<std::endl;
            }
            try {
                std::cout << std::endl << "################### ADAPTIVE MEMORY ##################" << std::endl;
                auto C = ModularityAlgorithms::Louvain(g, ADAPTIVE_MEMORY);
                std::cout << "N of community found:" << C.n_of_best_communities << std::endl ;

            }
            catch (std::bad_alloc e) {
                std::cout << "Bad Alloc" << std::endl;
            }

        }
        catch (std::bad_alloc e) {
            std::cout << "Bad Alloc Graph" << std::endl;
            continue;
        }
      
    }


  
    return 0;
}
