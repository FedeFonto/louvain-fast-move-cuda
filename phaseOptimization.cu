#include "hip/hip_runtime.h"
#ifndef OPTIMIZATION_CU     
#define OPTIMIZATION_CU

#include "phaseOptimization.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__
void select_self_node(
	unsigned int n_nodes,
	unsigned int* source,
	unsigned int* dest,
	unsigned int* community,
	float* value,
	float* self
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_nodes) {
		int node = source[id];
		if (community[node] == dest[id]) {
			self[node] = value[id];
		}
	}
};


__global__
void update_value_kernel(
	unsigned int n_nodes,
	unsigned int* nodes_to_update,
	unsigned int* community_dest,
	float* delta_value,
	unsigned int* community,
	float* community_weight,
	float* nodes_weight,
	bool* its_changed
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_nodes) {
		int node = nodes_to_update[id];
		int c = community_dest[id];
		if (community[node] == c || delta_value[id] <= 0) {
			return;
		}
		else {
			atomicAdd(&community_weight[c], nodes_weight[node]);
			atomicAdd(&community_weight[community[node]], nodes_weight[node] * -1);
			community[node] = c;
			its_changed[node] = true;
		}
	}
};

__global__
void update_changed_kernel(
	bool* n_changed,
	bool* its_changed,
	unsigned int* source,
	unsigned int* dest,
	unsigned int n_edge
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_edge) {
		if (its_changed[dest[id]]) {
			n_changed[source[id]] = true;
		}
	}
}

void OptimizationPhase::optimize() {
	auto its_changed = thrust::device_vector<bool>(community.graph.n_nodes, false);

#if PRINT_PERFORMANCE_LOG
		hipEvent_t start, copy, sort, reduce_sort, self_c, transform, reduce_transform, kernel_update, stop;
		hipEventCreate(&start);
		hipEventCreate(&copy);
		hipEventCreate(&sort);
		hipEventCreate(&reduce_sort);
		hipEventCreate(&self_c);
		hipEventCreate(&transform);
		hipEventCreate(&reduce_transform);
		hipEventCreate(&kernel_update);
		hipEventCreate(&stop);
		hipEventRecord(start);
#endif

	auto key_community_source = thrust::device_vector<unsigned int>(community.graph.edge_source.size());
	auto key_community_dest = thrust::device_vector<unsigned int>(community.graph.edge_source.size());
	auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_community_source.begin(), key_community_dest.begin()));


	auto values_weight = thrust::device_vector<float>(community.graph.edge_source);
	auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_community_source.begin(), key_community_dest.begin(), values_weight.begin()));

	auto p = thrust::copy_if(
		thrust::make_transform_iterator(community.start, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
		thrust::make_transform_iterator(community.end, MakeCommunityDest(thrust::raw_pointer_cast(community.communities.data()))),
		thrust::make_zip_iterator(thrust::make_tuple(community.graph.edge_source.begin(), community.graph.edge_destination.begin())),
		selected_edge,
		TestTupleValue(thrust::raw_pointer_cast(neighboorhood_change.data()))
	);

#if PRINT_PERFORMANCE_LOG
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, copy);
		std::cout << " - Copy Time : " << milliseconds << "ms" << std::endl;
#endif

	int n_edge_in_buckets = p - selected_edge;
	key_community_source.resize(n_edge_in_buckets);
	key_community_dest.resize(n_edge_in_buckets);
	values_weight.resize(n_edge_in_buckets);

	if (n_edge_in_buckets == 0) {
#if PRINT_DEBUG_LOG
		printf("No elements!\n");
#endif 
		return;
	}

	thrust::sort_by_key(
		key_community,
		key_community + n_edge_in_buckets,
		values_weight.begin()
	);

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(sort);
	hipEventSynchronize(sort);
	hipEventElapsedTime(&milliseconds, copy, sort);
	std::cout << " - Sort Time : " << milliseconds << "ms" << std::endl;
#endif

	auto reduced_key_source = thrust::device_vector<unsigned int>(n_edge_in_buckets);
	auto reduced_key_dest = thrust::device_vector<unsigned int>(n_edge_in_buckets);
	auto reduced_value = thrust::device_vector<float>(n_edge_in_buckets);

	auto reduced_key = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin()));
	auto reduced_list = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_source.begin(), reduced_key_dest.begin(), reduced_value.begin()));


	auto new_end = thrust::reduce_by_key(
		key_community,
		key_community + n_edge_in_buckets,
		values_weight.begin(),
		reduced_key,
		reduced_value.begin()
	);

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(reduce_sort);
	hipEventSynchronize(reduce_sort);
	+hipEventElapsedTime(&milliseconds, sort, reduce_sort);
	std::cout << " - Reduce after sort Time : " << milliseconds << "ms" << std::endl;
#endif

	auto n_reduced_edges = new_end.first - reduced_key;
	reduced_key_source.resize(n_reduced_edges);
	reduced_key_dest.resize(n_reduced_edges);
	reduced_value.resize(n_reduced_edges);

	auto self_community = thrust::device_vector<float>(community.graph.n_nodes);

	int n_blocks = (n_reduced_edges + BLOCK_SIZE - 1) / BLOCK_SIZE;


	select_self_node <<<n_blocks, BLOCK_SIZE >>> (
		n_reduced_edges,
		thrust::raw_pointer_cast(reduced_key_source.data()),
		thrust::raw_pointer_cast(reduced_key_dest.data()),
		thrust::raw_pointer_cast(community.communities.data()),
		thrust::raw_pointer_cast(reduced_value.data()),
		thrust::raw_pointer_cast(self_community.data())
	);		
			
#if PRINT_PERFORMANCE_LOG
	hipEventRecord(self_c);
	hipEventSynchronize(self_c);
	hipEventElapsedTime(&milliseconds, reduce_sort, self_c);
	std::cout << " - Reduce after transform Time : " << milliseconds << "ms" << std::endl;
#endif

	thrust::transform(
		reduced_list,
		reduced_list + n_reduced_edges,
		reduced_value.begin(),
		DeltaModularity(
			thrust::raw_pointer_cast(community.communities_weight.data()),
			thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
			community.graph.total_weight,
			thrust::raw_pointer_cast(self_community.data()),
			thrust::raw_pointer_cast(community.communities.data())
		)
	);

	/*for (int i = 0; i < n_reduced_edges; i++) {
		std::cout << reduced_key_source[i] << " " << community.communities[reduced_key_source[i]] << " " << reduced_key_dest[i] << " " << reduced_value[i] << std::endl;
	}*/

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(transform);
	hipEventSynchronize(transform);
	hipEventElapsedTime(&milliseconds, self_c, transform);
	std::cout << " - Transform Time : " << milliseconds << "ms" << std::endl;
#endif

		

	auto community_value_pair_input = thrust::make_zip_iterator(thrust::make_tuple(reduced_key_dest.begin(), reduced_value.begin()));
	auto community_value_pair_output = thrust::make_zip_iterator(thrust::make_tuple(key_community_dest.begin(), values_weight.begin()));

	auto ne = thrust::reduce_by_key(
		reduced_key_source.begin(),
		reduced_key_source.begin() + n_reduced_edges,
		community_value_pair_input,
		key_community_source.begin(),
		community_value_pair_output,
		thrust::equal_to<int>(),
		GetMaxValue()
	);

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(reduce_transform);
	hipEventSynchronize(reduce_transform);
	hipEventElapsedTime(&milliseconds, transform, reduce_transform);
	std::cout << " - Reduce after transform Time : " << milliseconds << "ms" << std::endl;
#endif

	n_reduced_edges = ne.first - key_community_source.begin();
	n_blocks = (n_reduced_edges + BLOCK_SIZE - 1) / BLOCK_SIZE;

	update_value_kernel << <n_blocks, BLOCK_SIZE >> > (
		n_reduced_edges,
		thrust::raw_pointer_cast(key_community_source.data()),
		thrust::raw_pointer_cast(key_community_dest.data()),
		thrust::raw_pointer_cast(values_weight.data()),
		thrust::raw_pointer_cast(community.communities.data()),
		thrust::raw_pointer_cast(community.communities_weight.data()),
		thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
		thrust::raw_pointer_cast(its_changed.data())
		);

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(kernel_update);
	hipEventSynchronize(kernel_update);
	hipEventElapsedTime(&milliseconds, reduce_transform, kernel_update);
	std::cout << " - Kernel Update Time : " << milliseconds << "ms" << std::endl;
#endif


	n_blocks = (community.graph.edge_source.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;

	thrust::fill(neighboorhood_change.begin(), neighboorhood_change.end(), false);

	update_changed_kernel << <n_blocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(neighboorhood_change.data()),
		thrust::raw_pointer_cast(its_changed.data()),
		thrust::raw_pointer_cast(community.graph.edge_source.data()),
		thrust::raw_pointer_cast(community.graph.edge_destination.data()),
		community.graph.edge_source.size()
	);


#if PRINT_PERFORMANCE_LOG
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, kernel_update, stop);
	std::cout << " - Kernel Changed Time : " << milliseconds << "ms" << std::endl;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Total Optimization Time: " << milliseconds << "ms \n" << std::endl;
#endif
}



#endif