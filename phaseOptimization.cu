#include "hip/hip_runtime.h"
#ifndef OPTIMIZATION_CU     
#define OPTIMIZATION_CU

#include "phaseOptimization.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__device__ __forceinline__ 
float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__global__
void update_best_kernel(unsigned long long* key, float* value, unsigned int* best_community, float* best_values, int n_edges){
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_edges) {
		const unsigned int node = key[id] >> 32;
		const unsigned int community = key[id];
		if (atomicMaxFloat(&best_values[node], value[id]) < value[id]) {
			atomicExch(&best_community[node], community);
		}
	}
}

__global__
void update_value_kernel(
	unsigned int n_nodes,
	unsigned int* community_dest,
	float* delta_value,
	unsigned int* community,
	float* community_weight,
	float* nodes_weight,
	bool* its_changed
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_nodes) {
		int node = id;
		int c = community_dest[id];
		if (community[node] == c || delta_value[id] <= 0) {
			return;
		}
		else {
			atomicAdd(&community_weight[c], nodes_weight[node]);
			atomicAdd(&community_weight[community[node]], nodes_weight[node] * -1);
			community[node] = c;
			its_changed[node] = true;
		}
	}
};

__global__
void update_changed_kernel(
	bool* n_changed,
	bool* its_changed,
	unsigned int* source,
	unsigned int* dest,
	unsigned int n_edge
) {
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_edge) {
		if (its_changed[dest[id]]) {
			n_changed[source[id]] = true;
		}
	}
}

void OptimizationPhase::optimize() {
#if PRINT_PERFORMANCE_LOG
	hipEvent_t start, round_start, copy, map, resize, transform, best, kernel_update, stop;
	hipEventCreate(&round_start);
	hipEventCreate(&start);
	hipEventCreate(&copy);
	hipEventCreate(&map);
	hipEventCreate(&resize);
	hipEventCreate(&transform);
	hipEventCreate(&best);
	hipEventCreate(&kernel_update);
	hipEventCreate(&stop);
	hipEventRecord(start);
	float milliseconds = 0;
#endif
	auto its_changed = thrust::device_vector<bool>(community.graph.n_nodes, false);

	int limit_round;
	int round = 0;

	while (round < community.graph.edge_destination.size()) {
#if PRINT_PERFORMANCE_LOG

		hipEventRecord(round_start);
#endif
		key_node_source.resize(STEP_ROUND);
		key_community_dest.resize(STEP_ROUND);
		values_weight.resize(STEP_ROUND);

		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		auto key_community = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin()));
		auto selected_edge = thrust::make_zip_iterator(thrust::make_tuple(key_node_source.begin(), key_community_dest.begin(), values_weight.begin()));

		int n_edge_in_buckets = limit_round;

#if PRINT_PERFORMANCE_LOG
		hipEventRecord(copy);
		hipEventSynchronize(copy);
		hipEventElapsedTime(&milliseconds, round_start, copy);
#if CSV_FORM
		std::cout << n_edge_in_buckets << "," << community.graph.edge_source.size() << "," << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << ",";
#else

		std::cout << "\nNumber of Edges selected: " << n_edge_in_buckets << " / " << community.graph.edge_source.size() << " (" << (float)n_edge_in_buckets / community.graph.edge_source.size() * 100 << " %)" << std::endl;
		std::cout << " - Copy Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
		h.fill_for_optimization(	
				thrust::raw_pointer_cast(community.graph.edge_source.data()),
				thrust::raw_pointer_cast(community.graph.edge_destination.data()),
				thrust::raw_pointer_cast(community.graph.weights.data()),
				round,
				n_edge_in_buckets,
				community.graph.n_nodes,
				thrust::raw_pointer_cast(community.communities.data()), 
				thrust::raw_pointer_cast(neighboorhood_change.data())
			);

#if PRINT_PERFORMANCE_LOG
		hipEventRecord(map);
		hipEventSynchronize(map);
		hipEventElapsedTime(&milliseconds, copy, map);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else

		std::cout << " - Hashmap Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		n_edge_in_buckets = h.resize();


#if PRINT_PERFORMANCE_LOG
		hipEventRecord(resize);
		hipEventSynchronize(resize);
		hipEventElapsedTime(&milliseconds, map, resize);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else

		std::cout << " - Resize Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		auto pair = thrust::make_zip_iterator(thrust::make_tuple(h.key.begin(), h.values.begin()));
		
		thrust::transform(
			pair,
			pair + n_edge_in_buckets,
			h.values.begin(),
			DeltaModularity(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(h.self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);



#if PRINT_PERFORMANCE_LOG
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, resize, transform);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else

		std::cout << " - Delta Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		int n_blocks = (n_edge_in_buckets + BLOCK_SIZE - 1) / BLOCK_SIZE;
		update_best_kernel << <n_blocks, BLOCK_SIZE >> > (
			h.pointer_k,
			h.pointer_v,
			thrust::raw_pointer_cast(final_community.data()),
			thrust::raw_pointer_cast(final_value.data()),
			n_edge_in_buckets
			);

#if PRINT_PERFORMANCE_LOG
		hipEventRecord(best);
		hipEventSynchronize(best);
		hipEventElapsedTime(&milliseconds, transform, best);
#if CSV_FORM
		std::cout << milliseconds <<std::endl;
#else

		std::cout << " - Best Selection Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
		round = limit_round;
	}

	int n_blocks = (community.graph.n_nodes + BLOCK_SIZE - 1) / BLOCK_SIZE;


	update_value_kernel << <n_blocks, BLOCK_SIZE >> > (
		community.graph.n_nodes,
		thrust::raw_pointer_cast(final_community.data()),
		thrust::raw_pointer_cast(final_value.data()),
		thrust::raw_pointer_cast(community.communities.data()),
		thrust::raw_pointer_cast(community.communities_weight.data()),
		thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
		thrust::raw_pointer_cast(its_changed.data())
		);

#if PRINT_PERFORMANCE_LOG
	hipEventRecord(kernel_update);
	hipEventSynchronize(kernel_update);
	hipEventElapsedTime(&milliseconds, best, kernel_update);
#if CSV_FORM
	std::cout << milliseconds << ",";
#else
	std::cout << " - Kernel Update Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

	n_blocks = (community.graph.edge_source.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;

	thrust::fill(neighboorhood_change.begin(), neighboorhood_change.end(), false);


	update_changed_kernel << <n_blocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(neighboorhood_change.data()),
		thrust::raw_pointer_cast(its_changed.data()),
		thrust::raw_pointer_cast(community.graph.edge_source.data()),
		thrust::raw_pointer_cast(community.graph.edge_destination.data()),
		community.graph.edge_source.size()
		);


#if PRINT_PERFORMANCE_LOG
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, kernel_update, stop);
#if CSV_FORM
	std::cout << milliseconds << ",";
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << milliseconds << std::endl;

#else
	std::cout << " - Kernel Changed Time : " << milliseconds << "ms" << std::endl;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Optimization Time: " << milliseconds << "ms \n" << std::endl;
#endif
#endif
}



#endif