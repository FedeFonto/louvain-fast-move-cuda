#include "hip/hip_runtime.h"
#ifndef OPTIMIZATION_CU     
#define OPTIMIZATION_CU

#include "phaseOptimization.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__device__ __forceinline__ 
float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}
__device__ __host__
unsigned int float_to_comparable_integer(float f) {
	return *reinterpret_cast<unsigned int*>(&f);
}


__global__
void update_best_kernel(unsigned long long* key, float* value, unsigned long long int* best_community, int n_edges){
	int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (id < n_edges) {
		unsigned int community = key[id];
		unsigned int node = key[id] >> 32;
		if (value[id] > 0) {
			auto k = (((unsigned long long) float_to_comparable_integer(value[id])) << 32 | community);
			atomicMax(&best_community[node], k);
		}
		
	}
}


void OptimizationPhase::optimize_hash() {
#if PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
	hipEvent_t  round_start, map, resize, transform, best, kernel_update, stop;
	hipEventCreate(&round_start);
	hipEventCreate(&map);
	hipEventCreate(&resize);
	hipEventCreate(&transform);
	hipEventCreate(&best);
	hipEventCreate(&kernel_update);
	hipEventCreate(&stop);
	float milliseconds = 0;
#endif

	int limit_round;
	int round = 0;

	while (round < community.graph.edge_destination.size()) {
#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(round_start);
#endif
	
		limit_round = round + STEP_ROUND;
		if (limit_round >= community.graph.edge_destination.size()) {
			limit_round = community.graph.edge_destination.size();
		}
		else {
			limit_round = community.graph.neighboorhood_sum[community.graph.edge_source[limit_round] - 1];
		}

		hashmap->fill_for_optimization(	
				thrust::raw_pointer_cast(community.graph.edge_source.data()),
				thrust::raw_pointer_cast(community.graph.edge_destination.data()),
				thrust::raw_pointer_cast(community.graph.weights.data()),
				round,
				limit_round,
				community.graph.n_nodes,
				thrust::raw_pointer_cast(community.communities.data()), 
				thrust::raw_pointer_cast(neighboorhood_change.data())
			);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(map);
		hipEventSynchronize(map);
		hipEventElapsedTime(&milliseconds, round_start, map);
#if CSV_FORM
		std::cout << limit_round - round - hashmap->conflict_stats[3] << "," << community.graph.edge_source.size() << "," << (float)(limit_round - round - hashmap->conflict_stats[3]) / community.graph.edge_source.size() * 100 << ",";
		std::cout <<(float) hashmap->conflict_stats[1] /  hashmap->conflict_stats[0] << "," ;
		std::cout << milliseconds << ",";
#else

		std::cout << "\nNumber of Edges selected: " << limit_round - round - hashmap->conflict_stats[3] << " / " << community.graph.edge_source.size() << " (" << (float)(limit_round - round - hashmap->conflict_stats[3]) / community.graph.edge_source.size() * 100 << " %)" << std::endl;
		std::cout << "Number of mean addressing: " <<(float) hashmap->conflict_stats[1] / (limit_round - round - hashmap->conflict_stats[3]) << std::endl;

		std::cout << " - Hashmap Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		int n_edge_in_buckets = hashmap->contract_array();


#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(resize);
		hipEventSynchronize(resize);		
		hipEventElapsedTime(&milliseconds, map, resize);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else

		std::cout << " - Resize Time : " << milliseconds << "ms" << std::endl;
#endif
#endif

		auto pair = thrust::make_zip_iterator(thrust::make_tuple(hashmap->key.begin(), hashmap->values.begin()));
		
		thrust::transform(
			pair,
			pair + n_edge_in_buckets,
			hashmap->values.begin(),
			DeltaModularityHash(
				thrust::raw_pointer_cast(community.communities_weight.data()),
				thrust::raw_pointer_cast(community.graph.tot_weight_per_nodes.data()),
				community.graph.total_weight,
				thrust::raw_pointer_cast(hashmap->self_community.data()),
				thrust::raw_pointer_cast(community.communities.data())
			)
		);



#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(transform);
		hipEventSynchronize(transform);
		hipEventElapsedTime(&milliseconds, resize, transform);
#if CSV_FORM
		std::cout << milliseconds << ",";
#else

		std::cout << " - Delta Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
	
		thrust::max_element(hashmap->values.begin(), hashmap->values.begin() + n_edge_in_buckets);
		int n_blocks = (n_edge_in_buckets + BLOCK_SIZE - 1) / BLOCK_SIZE;
		update_best_kernel << <n_blocks, BLOCK_SIZE >> > (
			hashmap-> pointer_k,
			hashmap-> pointer_v,
			thrust::raw_pointer_cast(final_pair.data()),
			n_edge_in_buckets
			);

#if  PRINT_PERFORMANCE_LOG && INCLUDE_SUBPHASE
		hipEventRecord(best);
		hipEventSynchronize(best);
		hipEventElapsedTime(&milliseconds, transform, best);
#if CSV_FORM
		std::cout << milliseconds <<std::endl;
#else

		std::cout << " - Best Selection Time : " << milliseconds << "ms" << std::endl;
#endif
#endif
		round = limit_round;
	}

}



#endif